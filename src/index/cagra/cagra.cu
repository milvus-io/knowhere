#include "hip/hip_runtime.h"
#include "cagra_config.h"
#include "common/raft/res_pool.cuh"
#include "common/raft_metric.h"
#include "knowhere/index_node.h"
#include "knowhere/log.h"
#include "raft/neighbors/cagra.cuh"
#include "raft/neighbors/cagra_serialize.cuh"
namespace knowhere {

using idx_type = uint32_t;

using cagra_index = raft::neighbors::experimental::cagra::index<float, idx_type>;

namespace detail {
struct device_setter {
    device_setter(int new_device)
        : prev_device_{[]() {
              auto result = int{};
              RAFT_CUDA_TRY(hipGetDevice(&result));
              return result;
          }()} {
        RAFT_CUDA_TRY(hipSetDevice(new_device));
    }

    ~device_setter() {
        RAFT_CUDA_TRY_NO_THROW(hipSetDevice(prev_device_));
    }

 private:
    int prev_device_;
};
}  // namespace detail
class CagraIndexNode : public IndexNode {
 public:
    CagraIndexNode(const Object& object) : devs_{}, gpu_index_{} {
    }

    virtual Status
    Build(const DataSet& dataset, const Config& cfg) override {
        auto err = Train(dataset, cfg);
        if (err != Status::success)
            return err;
        return Status::success;
    }

    virtual Status
    Train(const DataSet& dataset, const Config& cfg) override {
        auto cagra_cfg = static_cast<const knowhere::CagraConfig&>(cfg);
        if (gpu_index_) {
            LOG_KNOWHERE_WARNING_ << "index is already trained";
            return Status::index_already_trained;
        }
        if (cagra_cfg.gpu_ids.size() != 1) {
            LOG_KNOWHERE_WARNING_ << "Cagra implementation is single-GPU only" << std::endl;
            return Status::raft_inner_error;
        }
        auto metric = Str2RaftMetricType(cagra_cfg.metric_type);
        if (!metric.has_value()) {
            LOG_KNOWHERE_WARNING_ << "please check metric value: " << cagra_cfg.metric_type;
            return metric.error();
        }
        if (metric.value() != raft::distance::DistanceType::L2Expanded) {
            LOG_KNOWHERE_WARNING_ << "only support L2Expanded metric type";
            return Status::invalid_metric_type;
        }
        devs_.insert(devs_.begin(), cagra_cfg.gpu_ids.begin(), cagra_cfg.gpu_ids.end());
        auto scoped_device = detail::device_setter{*cagra_cfg.gpu_ids.begin()};
        auto build_params = raft::neighbors::experimental::cagra::index_params{};
        build_params.intermediate_graph_degree = cagra_cfg.intermediate_graph_degree;
        build_params.graph_degree = cagra_cfg.graph_degree;
        build_params.metric = metric.value();
        thread_local rmm::hip_stream stream;
        thread_local rmm::mr::cuda_memory_resource mr;
        thread_local raft::device_resources res(stream.view(), nullptr, &mr);
        auto rows = dataset.GetRows();
        auto dim = dataset.GetDim();
        auto* data = reinterpret_cast<float const*>(dataset.GetTensor());
        auto data_gpu = raft::make_device_matrix<float, idx_type>(res, rows, dim);
        RAFT_CUDA_TRY(hipMemcpyAsync(data_gpu.data_handle(), data, data_gpu.size() * sizeof(float), hipMemcpyDefault,
                                      stream.value()));
        gpu_index_ = raft::neighbors::experimental::cagra::build(
            res, build_params,
            raft::make_device_matrix_view<const float, idx_type>((const float*)data_gpu.data_handle(), rows, dim));
        this->dim_ = dim;
        this->counts_ = rows;
        res.sync_stream();
    }

    virtual Status
    Add(const DataSet& dataset, const Config& cfg) override {
        return Status::success;
    }

    virtual expected<DataSetPtr>
    Search(const DataSet& dataset, const Config& cfg, const BitsetView& bitset) const override {
        auto cagra_cfg = static_cast<const CagraConfig&>(cfg);
        auto rows = dataset.GetRows();
        auto dim = dataset.GetDim();
        auto* data = reinterpret_cast<float const*>(dataset.GetTensor());
        auto output_size = rows * cagra_cfg.k;
        auto ids = std::unique_ptr<idx_type[]>(new idx_type[output_size]);
        auto dis = std::unique_ptr<float[]>(new float[output_size]);
        try {
            auto scoped_device = detail::device_setter{devs_[0]};
            auto* res_ = &raft_res_pool::get_context().resources_;

            auto data_gpu = raft::make_device_matrix<float, idx_type>(*res_, rows, dim);
            raft::copy(data_gpu.data_handle(), data, data_gpu.size(), res_->get_stream());

            auto search_params = raft::neighbors::experimental::cagra::search_params{};
            search_params.max_queries = cagra_cfg.max_queries;
            auto ids_dev = raft::make_device_matrix<idx_type, idx_type>(*res_, rows, cagra_cfg.k);
            auto dis_dev = raft::make_device_matrix<float, idx_type>(*res_, rows, cagra_cfg.k);
            raft::neighbors::experimental::cagra::search(*res_, search_params, *gpu_index_,
                                                         raft::make_const_mdspan(data_gpu.view()), ids_dev.view(),
                                                         dis_dev.view());

            raft::copy(ids.get(), ids_dev.data_handle(), output_size, res_->get_stream());
            raft::copy(dis.get(), dis_dev.data_handle(), output_size, res_->get_stream());
            res_->sync_stream();

        } catch (std::exception& e) {
            LOG_KNOWHERE_WARNING_ << "RAFT inner error, " << e.what();
            return Status::raft_inner_error;
        }
        return Status::not_implemented;
        //        return GenResultDataSet(rows, cagra_cfg.k, ids.release(), dis.release());
    }

    expected<DataSetPtr>
    RangeSearch(const DataSet& dataset, const Config& cfg, const BitsetView& bitset) const override {
        return Status::not_implemented;
    }

    virtual expected<DataSetPtr>
    GetVectorByIds(const DataSet& dataset) const override {
        return Status::not_implemented;
    }

    virtual bool
    HasRawData(const std::string& metric_type) const override {
        return false;
    }

    expected<DataSetPtr>
    GetIndexMeta(const Config& cfg) const override {
        return Status::not_implemented;
    }

    virtual Status
    Serialize(BinarySet& binset) const override {
        if (!gpu_index_.has_value()) {
            LOG_KNOWHERE_ERROR_ << "Can not serialize empty RaftCagraIndex.";
            return Status::empty_index;
        }
        std::stringbuf buf;
        std::ostream os(&buf);
        os.write((char*)(&this->dim_), sizeof(this->dim_));
        os.write((char*)(&this->counts_), sizeof(this->counts_));
        os.write((char*)(&this->devs_[0]), sizeof(this->devs_[0]));

        auto scoped_device = detail::device_setter{devs_[0]};
        rmm::mr::cuda_memory_resource mr;
        rmm::hip_stream stm;
        raft::device_resources res(stm.view(), nullptr, &mr);

        raft::neighbors::experimental::cagra::serialize<float, idx_type>(res, os, *gpu_index_);

        os.flush();
        std::shared_ptr<uint8_t[]> index_binary(new (std::nothrow) uint8_t[buf.str().size()]);

        memcpy(index_binary.get(), buf.str().c_str(), buf.str().size());
        binset.Append(this->Type(), index_binary, buf.str().size());
        return Status::success;
    }

    virtual Status
    Deserialize(const BinarySet& binset, const Config& config) override {
        std::stringbuf buf;
        auto binary = binset.GetByName(this->Type());
        if (binary == nullptr) {
            LOG_KNOWHERE_ERROR_ << "Invalid binary set.";
            return Status::invalid_binary_set;
        }
        buf.sputn((char*)binary->data.get(), binary->size);
        std::istream is(&buf);

        is.read((char*)(&this->dim_), sizeof(this->dim_));
        is.read((char*)(&this->counts_), sizeof(this->counts_));
        this->devs_.resize(1);
        is.read((char*)(&this->devs_[0]), sizeof(this->devs_[0]));
        auto scoped_device = detail::device_setter{devs_[0]};

        thread_local rmm::mr::cuda_memory_resource mr;
        thread_local rmm::hip_stream stream;
        thread_local raft::device_resources res(stream.view(), nullptr, &mr);

        cagra_index index_ = raft::neighbors::experimental::cagra::deserialize<float, idx_type>(res, is);
        is.sync();
        gpu_index_ = cagra_index(std::move(index_));

        return Status::success;
    }

    virtual Status
    DeserializeFromFile(const std::string& filename, const Config& config) override {
    }

    virtual std::unique_ptr<BaseConfig>
    CreateConfig() const override {
        return std::make_unique<CagraConfig>();
    }

    virtual int64_t
    Dim() const override {
        return dim_;
    }

    virtual int64_t
    Size() const override {
        return 0;
    }

    virtual int64_t
    Count() const override {
        return counts_;
    }

    virtual std::string
    Type() const override {
        return knowhere::IndexEnum::INDEX_RAFT_IVFFLAT;
    }

 private:
    std::vector<int32_t> devs_;
    int64_t dim_ = 0;
    int64_t counts_ = 0;
    std::optional<cagra_index> gpu_index_;
};

}  // namespace knowhere
